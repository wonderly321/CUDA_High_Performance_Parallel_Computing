#include "hip/hip_runtime.h"
#include "kernel.h"
#define TX 32
#define TY 32
#define LEN 5.f
#define TIME_STEP 0.005f
#define FINAL_TIME 10.f

__device__ float scale(int i, int w) {
	return 2 * LEN*((1.f*i) / w - 0.5f);
}
__device__ float f(float x, float y, float param, float sys) {
	if (sys == 1) return x - 2 * param*y;
	if (sys == 2) return -x + param * (1 - x * x)*y;
	else return -x - 2 * param*y;
}

//exciplit Euler solver
__device__ float2 euler(float x, float y, float dt, float tFinal,float param, float sys) {
	float dx = 0.f, dy = 0.f;
	for (float t = 0; t < tFinal; t += dt) {
		dx = dt * y;
		dy = dt * f(x, y, param, sys);
		x += dx;
		y += dy;
	}
	return make_float2(x, y);
}

__device__ unsigned char clip(int n) {
	return n > 255 ? 255 : (n < 0 ? 0 : n);
}

__global__ void stabImageKernel(uchar4 *d_out, int w, int h, float p,int s) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x;
	const int r = blockIdx.y * blockDim.y + threadIdx.y;
	if ((c >= w) || (r >= h)) return;
	const int i = r * w + c;
	const float x0 = scale(c, w);
	const float y0 = scale(r, h);
	const float dist_0 = sqrt(x0*x0 + y0 * y0);
	const float2 pos = euler(x0, y0, TIME_STEP, FINAL_TIME, p, s);
	const float dist_f = sqrt(pos.x*pos.x + pos.y*pos.y);
	const float dist_r = dist_f / dist_0;
	d_out[i].x = clip(dist_r*255);
	d_out[i].y = ((c == w/2)||(r == h/2))?255:0;
	d_out[i].z = clip((1/dist_r)*255);
	d_out[i].w = 255;
}

void kernelLauncher(uchar4 * d_out, int w, int h, float p, int s)
{
	const dim3 blockSize(TX, TY);
	const dim3 gridSize = dim3((w + TX - 1) / TX, (h + TY - 1) / TY);

	stabImageKernel << <gridSize, blockSize >> > (d_out, w, h, p, s);

}

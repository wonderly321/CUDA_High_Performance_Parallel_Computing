#include "hip/hip_runtime.h"
#include <cstdio>
#define	W 32
#define H 32
#define D 32
#define TX 8
#define TY 8
#define TZ 8

int divUp(int a, int b) {
	return (a + b - 1) / b;
}
__device__ float distance(int c, int r, int s, float3 pos) {
	return sqrtf((c - pos.x)*(c - pos.x) + (r - pos.y)*(r - pos.y) + (s - pos.z)*(s - pos.z));
}

__global__ void distanceKernel(float *d_out, int w, int h, int d, float3 pos) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x;
	const int r = blockIdx.y * blockDim.y + threadIdx.y;
	const int s = blockIdx.z * blockDim.z + threadIdx.z;
	const int i = c + r * w + s * w * h;
	if ((c >= w) || (r >= h) || (s >= d)) return;
	d_out[i] = distance(c, r, s, pos);
}

int main() {
	float *out = (float*)calloc(W*H*D, sizeof(float));
	float *d_out = 0;
	hipMalloc(&d_out, W*H*D * sizeof(float));
	const float3 pos = { 0.0f,0.0f,0.0f };
	const dim3 blockSize(TX, TY, TZ);
	const dim3 gridSize(divUp(W, TX), divUp(H, TY), divUp(D, TZ));
	distanceKernel << <gridSize, blockSize >> > (d_out, W, H, D, pos);
	hipMemcpy(out, d_out, W*H*D * sizeof(float), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < W*H*D; i++) {
		printf("%f\n", out[i]);
	}
	hipFree(d_out);
	free(out);
	return 0;
}
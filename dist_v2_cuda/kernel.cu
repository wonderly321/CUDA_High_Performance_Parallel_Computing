#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
//#include <time.h>
#define TPB 3200
#define M 100 //Number of times to do the data transfer

__device__ float distance(float x1, float x2) {
	return sqrt((x2 - x1)*(x2 - x1));
}
__global__ void distanceKernel(float *d_out, float *d_in, float ref) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}
void distanceArray(float *out, float *in, float ref, int len) {
	//Create event variables for timing.
	hipEvent_t startMemcpy, stopMemcpy;
	hipEvent_t startKernel, stopKernel;
	hipEventCreate(&startMemcpy);
	hipEventCreate(&stopMemcpy);
	hipEventCreate(&startKernel);
	hipEventCreate(&stopKernel);

	float *d_in = 0;
	float *d_out = 0;

	checkCudaErrors(hipMalloc(&d_in, len * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_out, len * sizeof(float)));
	// Record the event that "start the clock" on data transfer
	hipEventRecord(startMemcpy);
	//clock_t memcpyBegin = clock();
	//Copy input data from host to device M times 
	for (int i = 0; i < M; i++) {
		checkCudaErrors(hipMemcpy(d_in, in, len * sizeof(float), hipMemcpyHostToDevice));
	}
	//Record the event that "stop the clock" on data transfer
	hipEventRecord(stopMemcpy);
	//clock_t memcpyEnd = clock();
	//Record the event that "start the clock" on kernel execution
	hipEventRecord(startKernel);
	//clock_t kernelBegin = clock();
	distanceKernel<<< len/TPB, TPB >>> (d_out, d_in, ref);
	//Record the event that "stop the clock" on kernel execution
	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());

	hipEventRecord(stopKernel);
	//clock_t kernelEnd = clock();

	checkCudaErrors(hipMemcpy(out, d_out, len * sizeof(float), hipMemcpyDeviceToHost));
	
	////Compute time in seconds between clock count readings
	//double memcpyTime = ((double)(memcpyEnd - memcpyBegin)) / CLOCKS_PER_SEC;
	//double kernelTime = ((double)(kernelEnd - kernelBegin)) / CLOCKS_PER_SEC;

	//Ensure timed events have stopped.
	hipEventSynchronize(stopMemcpy);
	hipEventSynchronize(stopKernel);
	//Cnovert event records to time and output.
	float memcpyTimeInMs = 0;
	hipEventElapsedTime(&memcpyTimeInMs, startMemcpy, stopMemcpy);
	float kernelTimeInMs = 0;
	hipEventElapsedTime(&kernelTimeInMs, startKernel, stopKernel);
	printf("Kernel time (ms): %f\n", kernelTimeInMs);
	printf("Data transfer time (ms):%f\n", memcpyTimeInMs);

	checkCudaErrors(hipFree(d_in));
	checkCudaErrors(hipFree(d_out));
}
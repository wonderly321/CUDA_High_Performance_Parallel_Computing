#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cstdio>

#define TPB 32

__device__ float distance(float x1, float x2) {
	return sqrt((x2 - x1)*(x2 - x1));
}
__global__ void distanceKernel(float *d_out, float *d_in, float ref) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}
void distanceArray(float *out, float *in, float ref, int len) {
	float *d_in = 0;
	float *d_out = 0;

	hipMalloc(&d_in, len * sizeof(float));
	hipMalloc(&d_out, len * sizeof(float));

	hipMemcpy(d_in, in, len * sizeof(float), hipMemcpyHostToDevice);

	distanceKernel<<< len/TPB, TPB >>> (d_out, d_in, ref);

	hipMemcpy(out, d_out, len * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}
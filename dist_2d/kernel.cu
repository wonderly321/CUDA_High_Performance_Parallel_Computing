#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#define W 500
#define H 500
#define TX 32
#define TY 32

__device__ unsigned char clip(int n) {
	return n > 255 ? 255 : (n < 0 ? 0 : n);
}
__global__ void distanceKernel(uchar4 *d_out, int w, int h, int2 pos) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x;
	const int r = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = r * w + c;
	if ((c >= w) || (r >= h)) return;
	const int d = sqrtf((c - pos.x)*(c - pos.y)) + (r - pos.x)*(r - pos.y);
	const unsigned char intensity = clip(255 - d);
	d_out[i].x = intensity;
	d_out[i].y = intensity;
	d_out[i].z = 0;
	d_out[i].z = 255;
}

int main() {
	uchar4 *out = (uchar4*)calloc(W*H, sizeof(uchar4));
	uchar4 *d_out;
	hipMalloc(&d_out, W*H * sizeof(uchar4));

	const int2 pos = { 0.0f,0.0f };
	const dim3 blockSize(TX, TY);
	const int bx = (W + TX - 1) / TX;
	const int by = (W + TY - 1) / TY;
	const dim3 gridSize = dim3(bx, by);
	distanceKernel << < gridSize, blockSize >> > (d_out, W, H, pos);
	hipMemcpy(out, d_out, W*H * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_out);
	
	free(out);	

	return 0;
}
#include "hip/hip_runtime.h"
#include "kernel.h"
#define TPB 64

__global__ void ddKernel(float *d_out, const float *d_in, int size, float h) {
	const int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i >= size) return;
	//sin ���׵������޲��
	float a1 = (d_in[i + 1] - d_in[i]) / (h);
	float a2 = (d_in[i] - d_in[i - 1]) / (h);
	d_out[i] = a1 ;
}

void ddParallel(float *out, const float *in, int n, float h) {
	float *d_in = 0, *d_out = 0;
	hipMalloc(&d_in, n * sizeof(float));
	hipMalloc(&d_out, n * sizeof(float));
	hipMemcpy(d_in, in, n * sizeof(float), hipMemcpyHostToDevice);

	ddKernel << <(n + TPB - 1) / TPB, TPB >> > (d_out, d_in, n, h);

	hipMemcpy(out, d_out, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out);
 }